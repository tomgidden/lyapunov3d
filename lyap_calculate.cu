#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, cuda
#include <driver_functions.h>
#include <hip/hip_runtime.h>

// CUDA utilities and system includes from the CUDA SDK samples
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include "kernel.hpp"
#include "scene.hpp"
#include "params.hpp"

// Image and grid parameters
const unsigned int volumeWidth = 512;
const unsigned int volumeHeight = volumeWidth;
const unsigned int volumeDepth = volumeWidth;
const unsigned int blockSize = 8;
const dim3 blocks(volumeWidth / blockSize, volumeHeight / blockSize, volumeDepth / blockSize);
const dim3 threads(blockSize, blockSize, blockSize);

LyapParams *curP = &prm;
LyapCam *curC = &cam;
unsigned int curL = 0;

#if USE_LMINMAX
#define LMIN prm->lMin
#define LMAX prm->lMax
#else
#define LMIN 0.0
#define LMAX 4.0
#endif

// Device array of lyapunov exponents
float *cudaExps = 0;

// Device sequence array
Int *cudaSeq;


void cuda_load_sequence(unsigned char *seqStr)
{
    size_t actual_length;
    Int *seq;

    actual_length = scene_convert_sequence(&seq, seqStr);

    checkCudaErrors(hipMalloc(&cudaSeq, actual_length * sizeof(Int)));
    checkCudaErrors(hipMemcpy(cudaSeq, seq, actual_length * sizeof(Int), hipMemcpyHostToDevice));

    free(seq);
}

void render()
{
    params_init();

    cuda_load_sequence(sequence);

    size_t expsSize = sizeof(float) * volumeWidth * volumeHeight * volumeDepth;

    // Allocate points memory
    checkCudaErrors(hipMalloc(&cudaExps, expsSize));

    // call CUDA kernel, writing results to PBO
    //    for(int i = 0; i < passes; ++i) {
    //        void *dummy;
    kernel_calc_volume<<<blocks, threads>>>(cudaExps, prm, cudaSeq);
    //        hipMemcpyAsync(dummy, dummy, 1, hipMemcpyDeviceToDevice);
    //    }
    getLastCudaError("kernel failed");

    printf("Points size = %ld\n", expsSize);

    float *myExps = (float *)malloc(expsSize);
    printf("malloc'ed %p.\n", myExps);

    checkCudaErrors(hipMemcpy( myExps, cudaExps, expsSize, hipMemcpyDeviceToHost ));

    FILE *fp = fopen("exps.raw", "wb");
    fwrite(myExps, 1, expsSize, fp);
    fclose(fp);

    free(myExps);

    getLastCudaError("dump failed");
}

void cleanup()
{
    checkCudaErrors(hipFree(cudaExps));
    checkCudaErrors(hipFree(cudaSeq));
}

int choose_cuda_device(int argc, char **argv, bool use_gl)
{
    int result = 0;

    result = findCudaDevice(argc, (const char **)argv);

    return result;
}

int main(int argc, char **argv)
{
    // Use command-line specified CUDA device, otherwise use device with
    // highest Gflops/s
    choose_cuda_device(argc, argv, true);

    render();

    cleanup();

    exit(EXIT_SUCCESS);
}
